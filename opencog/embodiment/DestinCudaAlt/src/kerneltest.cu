#include<stdio.h>
#include<stdlib.h>
#include<math.h>

#include "macros.h"
#include "node.h"
#include "destin.h"

void KernelTest()
{
    Node *n;

    uint nb, ni, ns;
    uint *inputOffsets = NULL;
    float *input_dev, *belief_dev;

    float *input_host;

    // allocate node
    MALLOC( n, Node, 1 );
    CUDAMALLOC( (void **) &n->node_dev, sizeof(CudaNode) );

    // give a random belief and input dimensionality
    nb = rand() % 190 + 10;
    ni = rand() % 190 + 10;
    ns = nb + ni;

    // allocate input and belief for node
    CUDAMALLOC( (void **) &input_dev, sizeof(float) * ni );
    CUDAMALLOC( (void **) &belief_dev, sizeof(float) * nb );

    float * stat_mem;
    CUDAMALLOC( (void **)&stat_mem, sizeof(float) * NodeStatsSize(ni, nb, 0) );

    InitNode( 0, ni, nb, 0, 0.1, 0.01, 0.1, n, n->node_dev, inputOffsets, input_dev, belief_dev, stat_mem);

    // allocate input frame
    MALLOC( input_host, float, ni );

    uint i,j;
    for( i=0; i < ni; i++ )
    {
        input_host[i] = (float) rand() / RAND_MAX;
    }

    CUDAMEMCPY( input_dev, input_host, sizeof(float) * ni, hipMemcpyHostToDevice );

    dim3 blocksize( 1, nb );

    CalculateDistances<<< blocksize, ns, sizeof(float)*2*ns >>>( n->node_dev, NULL );
    //NormalizeBelief<<< 1, nb, sizeof(float)*2*nb >>>( n->node_dev );
    NormalizeBeliefGetWinner<<< 1, nb, sizeof(float)*4*nb >>>( n->node_dev );
    CopyNodeFromDevice( n );

    float delta;
    float *dist;

    dist = (float *) malloc(sizeof(float) * nb);

    float errSum = 0;
    float distSum = 0;
    float err;

    for( i=0; i < nb; i++ )
    {
        dist[i] = 0;
        for( j=0; j < ns; j++ )
        {
            if( j < ni )
            {
                delta = n->mu[i*ns+j] - input_host[j];
            }
            else
            {
                delta = (n->mu[i*ns+j] - n->pBelief[j - ni]) * 0.5;
            }

            dist[i] += delta * delta;
        }

        dist[i] = 1 / dist[i];

        distSum += dist[i];

        err = (dist[i] - n->beliefEuc[i]) * (dist[i] - n->beliefEuc[i]);
        errSum += err;

        printf("  %0.3f\n", err);
    }

    float beliefEucSum = 0;

    for( i=0; i < nb; i++ ) 
    {
        dist[i] /= distSum;
        beliefEucSum += n->beliefEuc[i];
    }

    printf("MSE: %0.2f\n", errSum / nb);

    free(dist);
 
    CUDAFREE( input_dev );
    CUDAFREE( belief_dev );
    CUDAFREE( stat_mem );
    DestroyNode( n );
}

int main()
{
    srand(time(NULL));
    uint i;
    for( i=0; i < 10000; i++ )
    {
        KernelTest();
    }
}
