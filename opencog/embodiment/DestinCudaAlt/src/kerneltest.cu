#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include "macros.h"
#include "node.h"
#include "destin.h"

#define NLAYERS 3

void GetMSE( Destin *d1, Destin *d2 )
{
    if( d1->nNodes != d2->nNodes )
    {
        fprintf(stderr, "networks differ in # nodes!\n");
        exit(1);
    }

    float muMSE, sigmaMSE, starvMSE, pBeliefMSE, diff;
    float muMSE_Sum, sigmaMSE_Sum, starvMSE_Sum, pBeliefMSE_Sum;

    uint n, i, j;


    Node *n1, *n2;
    muMSE_Sum = sigmaMSE_Sum = starvMSE_Sum = pBeliefMSE_Sum = 0;

    for( n=0; n < d1->nNodes; n++ )
    {
        n1 = &(d1->nodes_host[n]);
        n2 = &(d2->nodes_host[n]);

        muMSE = sigmaMSE = starvMSE = pBeliefMSE = 0;

        if( n1->nb != n2->nb || n1->ns != n2->ns )
        {
            fprintf(stderr, "node %d differ in dimensionality!\n", n);
            exit(1);
        }

        for( i=0; i < n1->nb; i++ )
        {
            diff = n1->pBelief[i] - n2->pBelief[i];
            //diff = n1->pBelief[i];

            pBeliefMSE += diff * diff;
        }

        for( i=0; i < n1->nb; i++ )
        {
            for( j=0; j < n1->ns; j++ )
            {
                diff = n1->mu[n1->nb*i+j] - n2->mu[n1->nb*i+j];
                //diff = n1->mu[n1->nb*i+j];
                muMSE += diff*diff;

                diff = n1->sigma[n1->nb*i+j] - n2->sigma[n1->nb*i+j];
                //diff = n1->sigma[n1->nb*i+j];
                sigmaMSE += diff*diff;
            }

            diff = n1->starv[i] - n2->starv[i];
            //diff = n1->starv[i];
            starvMSE += diff*diff;
        }

/*
        printf("pBelief mse: %0.20f\n", pBeliefMSE / n1->nb);
        printf("mu mse: %0.20f\n", muMSE / (n1->ns*n1->nb));
        printf("sigma mse: %0.20f\n", sigmaMSE / (n1->ns*n1->nb));
        printf("starv mse: %0.20f\n", starvMSE / (n1->nb));
*/

        pBeliefMSE_Sum += pBeliefMSE / n1->nb;
        muMSE_Sum += muMSE / (n1->ns * n1->nb);
        sigmaMSE_Sum += sigmaMSE / (n1->ns * n1->nb);
        starvMSE_Sum += starvMSE / n1->nb;
    }

    printf("pBelief mse: %0.20f\n", pBeliefMSE_Sum / d1->nNodes);
    printf("mu mse: %0.20f\n", muMSE_Sum / d1->nNodes);
    printf("sigma mse: %0.20f\n", sigmaMSE_Sum / d1->nNodes);
    printf("starv mse: %0.20f\n", starvMSE_Sum / d1->nNodes);

}

void GenerateFrame( float * frame_host, float * frame_dev, uint frameSize )
{
    uint i;
    for( i=0; i < frameSize; i++ )
    {
        frame_host[i] = (float) rand() / RAND_MAX;
    }

    CUDAMEMCPY( frame_dev, frame_host, sizeof(float) * frameSize, hipMemcpyHostToDevice );

    return;
}

int main()
{
    Destin *d_cuda, *d_cpu;
    uint i, nLayers, nIt;
    uint *dims = NULL;
    float *frame_host = NULL, *frame_dev = NULL;

    nLayers = 7;
    nIt = 1000;
    
    srand(0);
    MALLOC( dims, uint, nLayers );
    for( i=0; i < nLayers; i++ )
    {
        dims[i] = rand() % 20 + 20;
    }
    
    uint frameSize = 1;
    for( i=0; i < nLayers - 1; i++ )
    {
        frameSize *= 4;
    }

    frameSize *= 16;

    MALLOC( frame_host, float, frameSize );
    CUDAMALLOC( (void **) &frame_dev, sizeof(float) * frameSize );

    srand(0);
    d_cuda = InitDestin(16, nLayers, dims, 0);

    srand(0);
    d_cpu = InitDestin(16, nLayers, dims, 0);

    ClearBeliefs( d_cuda );
    ClearBeliefs( d_cpu );


    GenerateFrame( frame_host, frame_dev, frameSize );
    float cardStart = (float) clock() / CLOCKS_PER_SEC;
    // do nIt training iterations on card
    for( i=0; i < nIt; i++ )
    {
        FormulateBelief( d_cuda, true, frame_dev);
    }

    // do nIt feedforwards on card
    for( i=0; i < nIt; i++ )
    {
        FormulateBelief( d_cuda, false, frame_dev);
    }
    float cardStop = (float) clock() / CLOCKS_PER_SEC;
    
    float tCard = cardStop - cardStart;
    
    printf("Card: %0.3f\n", tCard);

    // do nIt training iterations on host
    float hostStart = (float) clock() / CLOCKS_PER_SEC;
    for( i=0; i < nIt; i++ )
    {
        __CPU_FormulateBelief( d_cpu, true, frame_host);
    }

    // do nIt feedforwards on host
    for( i=0; i < nIt; i++ )
    {
        __CPU_FormulateBelief( d_cpu, false, frame_host);
    }
    float hostStop = (float) clock() / (CLOCKS_PER_SEC);

    float tHost = hostStop - hostStart;
    printf("Host: %0.3f\n", tHost);

    printf("Speedup: %0.2fx\n", tHost / tCard);

    CopyDestinFromDevice( d_cuda );

    GetMSE( d_cpu, d_cuda );

/*
    for( i=0; i < nIt; i++ )
    {
        GenerateFrame( frame_host, frame_dev, frameSize );

        FormulateBelief( d_cuda, true, frame_dev);
        __CPU_FormulateBelief( d_cpu, true, frame_host);

        CopyDestinFromDevice( d_cuda );

        GetMSE( d_cpu, d_cuda );
    }
*/

    DestroyDestin( d_cpu );
    DestroyDestin( d_cuda );
    
    FREE( dims );
    FREE( frame_host );
    CUDAFREE( frame_dev );

    return 0;
}
