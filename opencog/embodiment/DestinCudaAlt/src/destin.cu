#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<math.h>

#include "macros.h"
#include "node.h"
#include "destin.h"

Destin * InitDestin( uint ni, uint nl, uint *nb, uint nMovements )
{
    uint nNodes, nInputPipeline;
    uint i, l, nBeliefs, maxNs, maxNb;
    size_t bOffset, iOffset;

    float alpha, beta, starvCoeff;

    Destin *d;

    // initialize a new Destin object
    MALLOC(d, Destin, 1);

    MALLOC(d->nBeliefsPerNode, int, nl);
    memcpy(d->nBeliefsPerNode, nb, nl*sizeof(int));

    d->nNodes = 0;
    d->nLayers = nl;

    d->nMovements = nMovements;

    // get number of nodes to allocate
    // starting from the top layer with one node,
    // each subsequent layer has 4x the nodes.
    //
    // eg., 2-layer: 05 nodes
    //      3-layer: 21 nodes
    //      4-layer: 85 nodes
    //
    // also keep track of the number of beliefs
    // to allocate

    MALLOC(d->layerSize, uint, nl);

    nNodes = 0;
    nBeliefs = 0;
    for( i=0, l=nl ; l != 0; l--, i++ )
    {
        d->layerSize[i] = 1 << 2*(l-1);
//        printf("layer[%d] has %d nodes\n", i, d->layerSize[i]);
        nNodes += d->layerSize[i];
        nBeliefs += d->layerSize[i] * nb[i];
    }

    d->nNodes = nNodes;

    // input pipeline -- all beliefs are copied from the output of each
    // node to the input of the next node on each timestep. we want
    // the belief of each node except for the top node (its output goes
    // to no input to another node) to be easily copied to the input
    // of the next node, so we allocate a static buffer for it.
    nInputPipeline = nBeliefs - nb[nl-1];

    d->nInputPipeline = nInputPipeline;

    // allocate node pointers on host and device
//    printf("allocating %d nodes\n", nNodes);
    MALLOC(d->nodes_host, Node, nNodes);
    CUDAMALLOC( (void **) &d->nodes_dev, sizeof(CudaNode)*nNodes);

    // allocate space for inputs to nodes
//    printf("allocating %d inputs\n", nInputPipeline);
    MALLOC(d->inputPipeline, float, sizeof(float)*nInputPipeline);
    CUDAMALLOC( (void **) &d->inputPipeline_dev, sizeof(float)*nInputPipeline);

    // allocate space for beliefs for nodes on host and device
//    printf("allocating %d beliefs\n", nBeliefs);
    MALLOC(d->belief, float, sizeof(float)*nBeliefs);
    CUDAMALLOC( (void **) &d->belief_dev, sizeof(float)*nBeliefs);

    d->nBeliefs = nBeliefs;

    alpha = 0.001;
    beta = 0.01;
    starvCoeff = 0.05;

    // init belief and input offsets (pointers to big belief and input chunks we
    // allocated above)
    bOffset = 0;
    iOffset = 0;

    // keep track of the max num of beliefs and states.  we need this information
    // to correctly call kernels later
    maxNb = 0;
    maxNs = 0;

    // allocate the input layer offsets.  each node gets an offset from
    // the frame it is presented with.  now computing it indirectly with
    // an array, but there's gotta be a closed-form way of getting the
    // input offset..
    //
    // **note** this is hard-coding a 4-to-1 reduction assuming visual
    // input (2d). we may want 2-to-1 reduction for audio input for
    // future research

    uint n, m;

    uint **inputOffsets;
    uint nInputNodes = pow(4,nl-1);

    MALLOC(inputOffsets, uint *, sizeof(uint **) * d->layerSize[0]);
    for( i=0; i < nInputNodes; i++ )
    {
        MALLOC(inputOffsets[i], uint, sizeof(uint) * nInputNodes);
    }

    // get integer sq root of layersize[0]
    uint layerSizeSqRoot = (uint) sqrt( d->layerSize[0] );

    // get integer sq root of ni for lowest layer.  asssumes input is a square.
    uint inputSizeSqRoot = (uint) sqrt( ni );


    // get column size of input image (assuming it is square)
    uint nc = (uint) sqrt( d->layerSize[0] * ni );

    // calculate offsets.
    uint a, b, innerIdx, bias;
    
    // iterate through rows... (nodes)
    for( i=0, m=0; m < layerSizeSqRoot; m+=2 )
    {
        // iterate through columns... (nodes)
        for( n=0; n < layerSizeSqRoot; n+=2, i+=4 )
        {
            // iterate through rows... (inputs)
            for( innerIdx = 0, a=0; a < inputSizeSqRoot; a++ )
            {
                // iterate through columns... (inputs)
                for( b=0; b < inputSizeSqRoot; b++, innerIdx++ )
                {
                    bias = m*nc*inputSizeSqRoot + n*inputSizeSqRoot;
                    inputOffsets[i+0][innerIdx] = bias + a*nc+b;
                    inputOffsets[i+1][innerIdx] = bias + a*nc+b+inputSizeSqRoot;
                    inputOffsets[i+2][innerIdx] = bias + (a+inputSizeSqRoot)*nc+b;
                    inputOffsets[i+3][innerIdx] = bias + (a+inputSizeSqRoot)*nc+b+inputSizeSqRoot;
                }
            }
        }
    }


    int layerSize;
    MALLOC( d->nodeRef, int *, sizeof(int **) * d->nLayers );

    // set up layer/row/col references for nodes
    for( i=0, l=0; l < d->nLayers - 1; l++ )
    {
        layerSize = d->layerSize[l];
        layerSizeSqRoot = (uint) sqrt( layerSize );

        MALLOC( d->nodeRef[l], int, sizeof(int *)*layerSize );

        for( m=0; m < layerSizeSqRoot; m+=2 )
        {
            for( n=0; n < layerSizeSqRoot; n+=2, i+=4 )
            {
                d->nodeRef[l][  m   * layerSizeSqRoot + n  ] = i;
                d->nodeRef[l][  m   * layerSizeSqRoot + n+1] = i+1;
                d->nodeRef[l][(m+1) * layerSizeSqRoot + n  ] = i+2;
                d->nodeRef[l][(m+1) * layerSizeSqRoot + n+1] = i+3;
            }
        }
    }

    // set up layer/row/col reference for top node
    MALLOC( d->nodeRef[d->nLayers - 1], int, sizeof(int *) * 1 );
    d->nodeRef[l][0] = d->nNodes - 1;

    //printf("\n");


    // create array of pointers, one pointer per layer. Each layer
    // gets a memory block to hold all the nodes statistics for all nodes
    // in the layer. pre allocate cuda memory for node statistics all at once
    // ahead of time because hipMalloc is very slow on some cards
    MALLOC( d->stats_dev, float *, nl );

    // allocate stats memory block for the input layer
    uint statsSizePerNode;
    statsSizePerNode = NodeStatsSize(ni, nb[0], nb[1]);
    CUDAMALLOC((void **)&d->stats_dev[0], sizeof(float) * statsSizePerNode * d->layerSize[0]);

    // allocate input offsets memory block for input layer
    CUDAMALLOC((void**)&d->inputOffsets_dev, sizeof(uint) * ni * d->layerSize[0]);

    // initialize zero-layer nodes
    float * statsOffset;
    uint * io_offset;
    for( n=0, i=0, statsOffset = d->stats_dev[0], io_offset = d->inputOffsets_dev ;
            i < d->layerSize[0];
            i++, n++, statsOffset += statsSizePerNode, io_offset += ni )
    {
        InitNode( n, ni, nb[0], nb[1],
                    starvCoeff, alpha, beta, 
                    &d->nodes_host[n], &d->nodes_dev[n], inputOffsets[n],
                    io_offset, NULL, &d->belief_dev[bOffset], statsOffset );

        // increment belief offset
        bOffset += nb[0];

    }

    // update max belief 
    if( nb[0] > maxNb )
    {
        maxNb = nb[0];
    }

    // update max state
    if( nb[0] + ni > maxNs )
    {
        maxNs = nb[1] + nb[0] + ni;
    }

    // initialize the rest of the network
    for( l=1; l < nl; l++ )
    {
        // update max belief
        if( nb[l] > maxNb )
        {
            maxNb = nb[l];
        }

        uint np = l == nl - 1 ? 0 : nb[l + 1];
        // update max state
        if( nb[l] + 4 * nb[l-1] + np > maxNs )
        {
            maxNs = nb[l] + 4 * nb[l-1] + np;
        }

        statsSizePerNode = NodeStatsSize(nb[l-1]*4, nb[l], np);
        CUDAMALLOC((void **)&d->stats_dev[l], sizeof(float) * statsSizePerNode * d->layerSize[l]);
        for( i=0, statsOffset = d->stats_dev[l] ; i < d->layerSize[l]; i++, n++, statsOffset+=statsSizePerNode )
        {
            InitNode( n, nb[l-1]*4, nb[l], np,
                    starvCoeff, alpha, beta,
                    &d->nodes_host[n], &d->nodes_dev[n],
                    NULL, NULL, &d->inputPipeline_dev[iOffset],
                    &d->belief_dev[bOffset], statsOffset );

            // increment previous belief offset (input to next node)
            iOffset += 4*nb[l-1];

            // increment belief offset (so beliefs are mapped contiguously in memory)
            bOffset += nb[l];
        }
    }
    
    LinkParentBeliefToChildren( d );

    // set up maximum state and belief sizes for kernel calling
    d->maxNs = maxNs;
    d->maxNb = maxNb;

    //printf("maxNb: %d. maxNs: %d\n", maxNb, maxNs);
    
    for( i=0; i < nInputNodes; i++ )
    {
        free(inputOffsets[i]);
    }

    free(inputOffsets);

    return d;
}

void LinkParentBeliefToChildren( Destin *d )
{
    CudaNode cudaNode_host, cudaNodeParent_host;

    Node *node, *parent;
    uint i, n, l;

    uint parentBias = d->layerSize[0];
    for( n=0, l=0; l < d->nLayers - 1; l++ )
    {
        for( i=0; i < d->layerSize[l]; i++, n++)
        {
            // get structs from device
            node = &d->nodes_host[n];
            parent = &d->nodes_host[parentBias + i / 4];

            // update values
            node->parent_pBelief = parent->pBelief;

            // get structs from card
            CUDAMEMCPY( &cudaNode_host, node->node_dev, sizeof(CudaNode), hipMemcpyDeviceToHost );
            CUDAMEMCPY( &cudaNodeParent_host, parent->node_dev, sizeof(CudaNode), hipMemcpyDeviceToHost );

            // update values
            cudaNode_host.parent_pBelief = cudaNodeParent_host.pBelief;

            // copy structs back to card
            CUDAMEMCPY( node->node_dev, &cudaNode_host, sizeof(CudaNode), hipMemcpyHostToDevice );
            CUDAMEMCPY( parent->node_dev, &cudaNodeParent_host, sizeof(CudaNode), hipMemcpyHostToDevice );
        }

        parentBias += d->layerSize[l+1];
    }

}

void PrintNetwork( Destin *d, float *inImg )
{
    Node *nPtr;

    CopyDestinFromDevice( d );

    uint i, j, l, n;

/*
    float avgClustErr;

    printf("avg clustering err:\n");
    for( n=0, l=0; l < d->nLayers; l++ )
    {
        avgClustErr = 0;

        for( i=0; i < d->layerSize[l]; i++, n++ )
        {
            avgClustErr += d->nodes_host[i].clustErr;
        }

        printf("    layer[%d]: %0.10f\n", l, avgClustErr / d->layerSize[l]);
    }
    printf("\n");
*/

    for( n=0, l=0; l < d->nLayers; l++ )
    {
        printf("layer %d:\n", l);

        for( i=0; i < d->layerSize[l]; i++, n++ )
        {
            nPtr = &d->nodes_host[n];

            printf("  node %d:\n", n, nPtr->ni);
            printf("INPUTS\n");
            if( nPtr->inputOffsets != NULL )
            {
                for( j=0; j < nPtr->ni; j++ )
                {
                    printf("%02.0f ", 100*inImg[ nPtr->inputOffsets[j] ] );
                }
            }
            else
            {
                for( j=0 ; j < nPtr->ni; j++ )
                {
                    printf("%02.0f ", 100*nPtr->input[j]);
                }
            }

            printf(" | ");

            for( j=0; j < nPtr->nb; j++ )
            {
                printf("%02.0f ", 100*nPtr->pBelief[j]);
            }
            
            printf(" | ");

            for( j=0; j < nPtr->np; j++ )
            {
                printf("%02.0f ", 100*nPtr->parent_pBelief[j]);
            }
            printf("\n\n");
            
            /*
            printf("STATISTICS\n");
            nPtr = &d->nodes_host[n];

            printf("  mu:\n");
            for( b=0; b < nPtr->nb; b++ )
            {
                printf("  ");
                for( s=0; s < nPtr->ns; s++ )
                {
                    printf("%0.2f ", nPtr->mu[s+b*nPtr->ns]);
                }
                printf("\n");
            }

            printf("\n\n");
            printf("  sigma:\n");
            for( b=0; b < nPtr->nb; b++ )
            {
                printf("  ");
                for( s=0; s < nPtr->ns; s++ )
                {
                    printf("%0.2f ", nPtr->sigma[s+b*nPtr->ns]);
                }
                printf("\n");
            }
            
            printf("\n\n");
            printf("  starvation:\n");
            printf("  ");
            for( b=0; b < nPtr->nb; b++ )
            {
                printf("%0.2f ", nPtr->starv[b]);
            }
            printf("\n");
            */
        }
    }
}

float * RunDestin( Destin *d, char *dataFileName, bool isTrain )
{
    // belief output.  allocate if isTrain is false (only output beliefs if the network
    // is trained)
    float *beliefOut = NULL;

    // check if destin passed is initialized
    if( d == NULL )
    {
        fprintf(stderr, "Destin 0x%p not initialized!\n", d);
        exit(1);
    }

    FILE *dataFile;
    
    // filesize in bytes
    size_t nFloats;
    
    dataFile = fopen(dataFileName, "r");
    if( !dataFile ) {
        fprintf(stderr, "Cannot open data file %s\n", dataFileName);
        exit(1);
    }

    // get filesize
    fseek(dataFile, 0L, SEEK_END);
    nFloats = ftell(dataFile) / 4; //TODO: should this divide by sizeof(float) instead?
    fseek(dataFile, 0L, SEEK_SET);

    // get remaining memory on card
    size_t deviceFree, deviceTotal;
    hipMemGetInfo(&deviceFree, &deviceTotal);

    size_t chunkSize;
    size_t inputFrameSize = d->layerSize[0] * 16; //TODO: replace magic number 16 with a constant variable

    uint nPresentations;

    if( !isTrain )
    {
        nPresentations = nFloats / inputFrameSize / d->nMovements;

        MALLOC( beliefOut, float, d->nBeliefs * nPresentations * 3 );
    }

    // if the dataset fits in 80% of remaining memory, allocate the whole she-bang
    if( nFloats < (size_t) ((float) (deviceFree/4) * 0.8) )
    {
        chunkSize = nFloats;
    }
    else
    {
        // otherwise, use ~80% of remaining memory for data chunks (rounding up to the next digit presentation size)
        chunkSize = (size_t) ((float) (deviceFree/4) * 0.8 );
        chunkSize += inputFrameSize*d->nMovements - (chunkSize % (inputFrameSize*d->nMovements));
    }

    // allocate space for data set
    MALLOC(d->dataSet, float, chunkSize);
    CUDAMALLOC( (void **) &d->dataSet_dev, sizeof(float)*chunkSize );

    size_t nFloatsRead = 0;

    uint i, iMod, wIt;

    wIt = 0;

    // while the whole file hasn't been read...
    while( nFloatsRead < nFloats )
    {
        size_t nFloats_it = 0;

        // read in a chunk
        i = 0;
        while( i < chunkSize )
        {
            if( feof( dataFile ) )
            {
                break;
            }
            nFloats_it += fread( &d->dataSet[i], sizeof(float), inputFrameSize, dataFile );

            i += inputFrameSize;
        }

        nFloatsRead += nFloats_it;

        // copy chunk of data set to device
        CUDAMEMCPY( d->dataSet_dev, d->dataSet, sizeof(float) * chunkSize, hipMemcpyHostToDevice );

        // get number of iterations to run
        uint nIt = nFloats_it / inputFrameSize;

        printf("presenting %d movements...\n", nIt);

        for( i=0; i < nIt; i++ )
        {
            // a new digit is picked up every d->nMovements movements.
            if( i % d->nMovements == 0 )
            {
                ClearBeliefs( d );
            }

            // formulate belief/update on the presentation
            FormulateBelief( d, isTrain, d->dataSet_dev + i*inputFrameSize );

            // write out beliefs if we aren't training
            if( !isTrain )
            {
                // copy out the 10,12,14 movements
                iMod = i % d->nMovements;
                if( iMod == 10 || iMod == 12 || iMod == 14 )
                {
                    CUDAMEMCPY( &beliefOut[wIt*d->nBeliefs], d->belief_dev, sizeof(float) * d->nBeliefs, hipMemcpyDeviceToHost );
                    wIt++;
                }
            }
        }
    }

    fclose( dataFile );

    FREE( d->dataSet );
    CUDAFREE( d->dataSet_dev );

    return beliefOut;
}

void CopyDestinToDevice( Destin *d )
{
    uint i;

    // copy individual nodes
    for( i=0; i < d->nNodes; i++ )
    {
        CopyNodeToDevice( &d->nodes_host[i] );
    }

    // copy belief and input information
    CUDAMEMCPY( d->belief_dev,        d->belief,        sizeof(float) * d->nBeliefs,       hipMemcpyHostToDevice );
    CUDAMEMCPY( d->inputPipeline_dev, d->inputPipeline, sizeof(float) * d->nInputPipeline, hipMemcpyHostToDevice );
}

void CopyDestinFromDevice( Destin *d )
{
    uint i;

    // copy individual nodes
    for( i=0; i < d->nNodes; i++)
    {
        CopyNodeFromDevice( &d->nodes_host[i] );
    }

    // copy belief and input information
    CUDAMEMCPY( d->belief,        d->belief_dev,        sizeof(float) * d->nBeliefs,       hipMemcpyDeviceToHost );
    CUDAMEMCPY( d->inputPipeline, d->inputPipeline_dev, sizeof(float) * d->nInputPipeline, hipMemcpyDeviceToHost );
}

void DestroyDestin( Destin *d )
{
    uint i;

    for( i=0; i < d->nNodes; i++ )
    {
        DestroyNode( &d->nodes_host[i] );
    }

    FREE(d->nodes_host);
    FREE(d->inputPipeline);
    FREE(d->belief);
    FREE(d->nBeliefsPerNode);
    FREE(d->layerSize);

    for( i=0; i < d->nLayers; i++ )
    {
        FREE( d->nodeRef[i] );
        CUDAFREE( d->stats_dev[i] );
    }

    FREE( d->nodeRef );
    FREE( d->stats_dev );

    CUDAFREE(d->nodes_dev);
    CUDAFREE(d->inputPipeline_dev);
    CUDAFREE(d->belief_dev);
    CUDAFREE(d->inputOffsets_dev);
    FREE(d);
}

// grab a node at a particular layer, row, and column
Node *GetNodeFromDestin( Destin *d, uint l, uint r, uint c )
{
    // check layer bounds
    if( l >= d->nLayers )
    {
        fprintf(stderr, "GetNodeFromDestin(): layer requested is out of range!\n");
        return NULL;
    }

    uint layerSizeSqRoot = (uint) sqrt( d->layerSize[l] );

    // check row bounds
    if( r >= layerSizeSqRoot )
    {
        fprintf(stderr, "GetNodeFromDestin(): row requested is out of range!\n");
        return NULL;
    }

    // check column bounds
    if( c >= layerSizeSqRoot )
    {
        fprintf(stderr, "GetNodeFromDestin(): column requested is out of range!\n");
        return NULL;
    }

    // grab the node index
    uint nIdx = d->nodeRef[l][r*layerSizeSqRoot+c];
    return &d->nodes_host[nIdx];
}

// present the network with an image pointed to by image_dev.  update the network if doUpdate is true,
// otherwise just change the belief.
void FormulateBelief(Destin *d, bool isTrain, float *image_dev)
{
    dim3 blockSize(d->nNodes, d->maxNb);

    CalculateDistances<<<blockSize,d->maxNs,sizeof(float)*d->maxNs*2>>>(d->nodes_dev, image_dev);
    if( isTrain )
    // normalize belief and update winner
    {
        NormalizeBeliefGetWinner<<<d->nNodes,d->maxNb,sizeof(float)*d->maxNb*4>>>(d->nodes_dev);
        UpdateWinner<<<d->nNodes,d->maxNs>>>(d->nodes_dev, image_dev);
    }
    else
    // just normalize the belief
    {
        NormalizeBelief<<<d->nNodes,d->maxNb,sizeof(float)*d->maxNb*2>>>(d->nodes_dev);
    }

    CUDAMEMCPY( d->inputPipeline_dev, d->belief_dev, sizeof(float)*d->nInputPipeline, hipMemcpyDeviceToDevice );

    CUDACHECKERROR();

    return;
}

// zeros-out the beliefs (and input registers) for each node.  useful for "restarting" the network
// between presentations of distinct elements
void ClearBeliefs( Destin *d )
{
    uint i;

    for( i=0; i < d->nInputPipeline; i++ )
    {
        d->inputPipeline[i] = 0;
    }

    for( i=0; i < d->nBeliefs; i++ )
    {
        d->belief[i] = 0;
    }
    
    // copy to card
    CUDAMEMCPY( d->inputPipeline_dev, d->inputPipeline, sizeof(float) * d->nInputPipeline, hipMemcpyHostToDevice);
    CUDAMEMCPY( d->belief_dev,        d->belief,        sizeof(float) * d->nBeliefs,       hipMemcpyHostToDevice);
}
