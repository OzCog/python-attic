#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<math.h>

#include "macros.h"
#include "node.h"
#include "destin.h"

Destin * InitDestin( int ni, int nl, int *nb, int nMovements )
{
    int nNodes, nInputPipeline;
    int i, l, nBeliefs, maxNs, maxNb;
    size_t bOffset, iOffset;

    float alpha, beta, starvCoeff;

    Destin *newDestin;

    // initialize a new Destin object
    MALLOC(newDestin, Destin, 1);

    newDestin->nNodes = 0;
    newDestin->nLayers = nl;

    newDestin->nMovements = nMovements;

    // get number of nodes to allocate
    // starting from the top layer with one node,
    // each subsequent layer has 4x the nodes.
    //
    // eg., 2-layer: 05 nodes
    //      3-layer: 21 nodes
    //      4-layer: 85 nodes
    //
    // also keep track of the number of beliefs
    // to allocate

    MALLOC(newDestin->layerSize, int, nl);

    nNodes = 0;
    nBeliefs = 0;
    for( i=0, l=nl-1 ; l >= 0; l--, i++ )
    {
        newDestin->layerSize[i] = 1 << 2*l;
        printf("layer[%d] has %d nodes\n", i, newDestin->layerSize[i]);
        nNodes += newDestin->layerSize[i];
        nBeliefs += newDestin->layerSize[i] * nb[i];
    }

    newDestin->nNodes = nNodes;

    // input pipeline -- all beliefs are copied from the output of each
    // node to the input of the next node on each timestep. we want
    // the belief of each node except for the top node (its output goes
    // to no input to another node) to be easily copied to the input
    // of the next node, so we allocate a static buffer for it.
    nInputPipeline = nBeliefs - nb[nl-1];

    newDestin->nInputPipeline = nInputPipeline;

    // allocate node pointers on host and device
    printf("allocating %d nodes\n", nNodes);
    MALLOC(newDestin->nodes_host, Node, nNodes);
    CUDAMALLOC( (void **) &newDestin->nodes_dev, sizeof(CudaNode)*nNodes);

    // allocate space for inputs to nodes
    printf("allocating %d inputs\n", nInputPipeline);
    MALLOC(newDestin->inputPipeline, float, sizeof(float)*nInputPipeline);
    CUDAMALLOC( (void **) &newDestin->inputPipeline_dev, sizeof(float)*nInputPipeline);

    // allocate space for beliefs for nodes on host and device
    printf("allocating %d beliefs\n", nBeliefs);
    MALLOC(newDestin->belief, float, sizeof(float)*nBeliefs);
    CUDAMALLOC( (void **) &newDestin->belief_dev, sizeof(float)*nBeliefs);

    newDestin->nBeliefs = nBeliefs;

    alpha = 0.001;
    beta = 0.01;
    starvCoeff = 0.05;

    // init belief and input offsets (pointers to big belief and input chunks we
    // allocated above)
    bOffset = 0;
    iOffset = 0;

    // keep track of the max num of beliefs and states.  we need this information
    // to correctly call kernels later
    maxNb = 0;
    maxNs = 0;

    // allocate the input layer offsets.  each node gets an offset from
    // the frame it is presented with.  now computing it indirectly with
    // an array, but there's gotta be a closed-form way of getting the
    // input offset..
    //
    // **note** this is hard-coding a 4-to-1 reduction assuming visual
    // input (2d). we may want 2-to-1 reduction for audio input for
    // future research

    int n, m;

    int **inputOffsets;
    int nInputNodes = pow(4,nl-1);

    MALLOC(inputOffsets, int *, sizeof(int **) * newDestin->layerSize[0]);
    for( i=0; i < nInputNodes; i++ )
    {
        MALLOC(inputOffsets[i], int, sizeof(int) * nInputNodes);
    }

    // get integer sq root of layersize[0]
    int layerSizeSqRoot = (int) sqrt( newDestin->layerSize[0] );

    // get integer sq root of ni for lowest layer.  asssumes input is a square.
    int inputSizeSqRoot = (int) sqrt( ni );


    // get column size of input image (assuming it is square)
    int nc = (int) sqrt( newDestin->layerSize[0] * ni );

    // calculate offsets.
    int a, b, innerIdx, bias;
    
    // iterate through rows... (nodes)
    for( i=0, m=0; m < layerSizeSqRoot; m+=2 )
    {
        // iterate through columns... (nodes)
        for( n=0; n < layerSizeSqRoot; n+=2, i+=4 )
        {
            // iterate through rows... (inputs)
            for( innerIdx = 0, a=0; a < inputSizeSqRoot; a++ )
            {
                // iterate through columns... (inputs)
                for( b=0; b < inputSizeSqRoot; b++, innerIdx++ )
                {
                    bias = m*nc*inputSizeSqRoot + n*inputSizeSqRoot;
                    inputOffsets[i+0][innerIdx] = bias + a*nc+b;
                    inputOffsets[i+1][innerIdx] = bias + a*nc+b+inputSizeSqRoot;
                    inputOffsets[i+2][innerIdx] = bias + (a+inputSizeSqRoot)*nc+b;
                    inputOffsets[i+3][innerIdx] = bias + (a+inputSizeSqRoot)*nc+b+inputSizeSqRoot;
                }
            }
        }
    }

    printf("\n");
    // initialize zero-layer nodes
    for( n=0, i=0; i < newDestin->layerSize[0]; i++, n++)
    {
        InitNode( n, ni, nb[0], nb[1],
                    starvCoeff, alpha, beta, 
                    &newDestin->nodes_host[n], &newDestin->nodes_dev[n],
                    inputOffsets[n], NULL, &newDestin->belief_dev[bOffset] );
                    

        // increment belief offset
        bOffset += nb[0];

    }

    // update max belief 
    if( nb[0] > maxNb )
    {
        maxNb = nb[0];
    }

    // update max state
    if( nb[0] + ni > maxNs )
    {
        maxNs = nb[1] + nb[0] + ni;
    }

    // initialize the rest of the network
    for( l=1; l < nl; l++ )
    {
        // update max belief
        if( nb[l] > maxNb )
        {
            maxNb = nb[l];
        }

        // update max state
        if( l == nl - 1 )
        {
            if( nb[l] + 4 * nb[l-1] > maxNs )
            {
                maxNs = nb[l] + 4 * nb[l-1];
            }
        }
        else
        {
            if( nb[l] + 4 * nb[l-1] + nb[l+1] > maxNs )
            {
                maxNs = nb[l] + 4 * nb[l-1] + nb[l+1];
            }
        }

        for( i=0; i < newDestin->layerSize[l]; i++, n++ )
        {
            if( l == nl - 1 )
            {
                InitNode( n, nb[l-1]*4, nb[l], 0,
                        starvCoeff, alpha, beta,
                        &newDestin->nodes_host[n], &newDestin->nodes_dev[n],
                        NULL, &newDestin->inputPipeline_dev[iOffset],
                        &newDestin->belief_dev[bOffset] );
            }
            else
            {
                InitNode( n, nb[l-1]*4, nb[l], nb[l+1],
                        starvCoeff, alpha, beta,
                        &newDestin->nodes_host[n], &newDestin->nodes_dev[n],
                        NULL, &newDestin->inputPipeline_dev[iOffset],
                        &newDestin->belief_dev[bOffset] );
            }
            // increment previous belief offset (input to next node)
            iOffset += 4*nb[l-1];

            // increment belief offset (so beliefs are mapped contiguously in memory)
            bOffset += nb[l];
        }
    }
    
    LinkParentBeliefToChildren( newDestin, nb );

    // set up maximum state and belief sizes for kernel calling
    newDestin->maxNs = maxNs;
    newDestin->maxNb = maxNb;

    printf("maxNb: %d. maxNs: %d\n", maxNb, maxNs);
    
    for( i=0; i < nInputNodes; i++ )
    {
        free(inputOffsets[i]);
    }

    free(inputOffsets);

    return newDestin;
}

void LinkParentBeliefToChildren( Destin *d, int *nb )
{
    CudaNode cudaNode_host, cudaNodeParent_host;

    Node *node, *parent;
    int i, n, l;

    int parentBias = d->layerSize[0];
    for( n=0, l=0; l < d->nLayers - 1; l++ )
    {
        for( i=0; i < d->layerSize[l]; i++, n++)
        {
            printf("linking child node %d to parent node %d\n", n, parentBias + i / 4);
            // get structs from device
            node = &d->nodes_host[n];
            parent = &d->nodes_host[parentBias + i / 4];

            // update values
            node->parent_pBelief = parent->pBelief;

            // get structs from card
            CUDAMEMCPY( &cudaNode_host, node->node_dev, sizeof(CudaNode), hipMemcpyDeviceToHost );
            CUDAMEMCPY( &cudaNodeParent_host, parent->node_dev, sizeof(CudaNode), hipMemcpyDeviceToHost );

            // update values
            cudaNode_host.parent_pBelief = cudaNodeParent_host.pBelief;

            // copy structs back to card
            CUDAMEMCPY( node->node_dev, &cudaNode_host, sizeof(CudaNode), hipMemcpyHostToDevice );
            CUDAMEMCPY( parent->node_dev, &cudaNodeParent_host, sizeof(CudaNode), hipMemcpyHostToDevice );
        }

        parentBias += d->layerSize[l+1];
    }

}

void PrintNetwork( Destin *d, float *inImg )
{
    Node *nPtr;

    CopyDestinFromDevice( d );

    int i, j, l, n;

/*
    float avgClustErr;

    printf("avg clustering err:\n");
    for( n=0, l=0; l < d->nLayers; l++ )
    {
        avgClustErr = 0;

        for( i=0; i < d->layerSize[l]; i++, n++ )
        {
            avgClustErr += d->nodes_host[i].clustErr;
        }

        printf("    layer[%d]: %0.10f\n", l, avgClustErr / d->layerSize[l]);
    }
    printf("\n");
*/

    for( n=0, l=0; l < d->nLayers; l++ )
    {
        printf("layer %d:\n", l);

        for( i=0; i < d->layerSize[l]; i++, n++ )
        {
            nPtr = &d->nodes_host[n];

            printf("  node %d:\n", n, nPtr->ni);
            printf("INPUTS\n");
            if( nPtr->inputOffsets != NULL )
            {
                for( j=0; j < nPtr->ni; j++ )
                {
                    printf("%02.0f ", 100*inImg[ nPtr->inputOffsets[j] ] );
                }
            }
            else
            {
                for( j=0 ; j < nPtr->ni; j++ )
                {
                    printf("%02.0f ", 100*nPtr->input[j]);
                }
            }

            printf(" | ");

            for( j=0; j < nPtr->nb; j++ )
            {
                printf("%02.0f ", 100*nPtr->pBelief[j]);
            }
            
            printf(" | ");

            for( j=0; j < nPtr->np; j++ )
            {
                printf("%02.0f ", 100*nPtr->parent_pBelief[j]);
            }
            printf("\n\n");
            
            /*
            printf("STATISTICS\n");
            nPtr = &d->nodes_host[n];

            printf("  mu:\n");
            for( b=0; b < nPtr->nb; b++ )
            {
                printf("  ");
                for( s=0; s < nPtr->ns; s++ )
                {
                    printf("%0.2f ", nPtr->mu[s+b*nPtr->ns]);
                }
                printf("\n");
            }

            printf("\n\n");
            printf("  sigma:\n");
            for( b=0; b < nPtr->nb; b++ )
            {
                printf("  ");
                for( s=0; s < nPtr->ns; s++ )
                {
                    printf("%0.2f ", nPtr->sigma[s+b*nPtr->ns]);
                }
                printf("\n");
            }
            
            printf("\n\n");
            printf("  starvation:\n");
            printf("  ");
            for( b=0; b < nPtr->nb; b++ )
            {
                printf("%0.2f ", nPtr->starv[b]);
            }
            printf("\n");
            */
        }
    }
}

void RunDestin( Destin *d, char *dataFileName, char *beliefFileName, bool isTrain )
{
    // check if destin passed is initialized
    if( d == NULL )
    {
        fprintf(stderr, "Destin 0x%p not initialized!\n", d);
        exit(1);
    }

    if( !isTrain && beliefFileName == NULL )
    {
        fprintf(stderr, "No belief filename given!\n");
        exit(1);
    }

    FILE *dataFile;
    FILE *beliefFile;
    
    // filesize in bytes
    size_t nFloats;
    
    dataFile = fopen(dataFileName, "r");
    if( !dataFile ) {
        fprintf(stderr, "Cannot open data file %s\n", dataFileName);
        exit(1);
    }

    if( !isTrain )
    {
        beliefFile = fopen(beliefFileName, "w");
        if( !beliefFile )
        {
            fprintf(stderr, "Cannot open belief file %s\n", beliefFileName);
            exit(1);
        }
    }

    // get filesize
    fseek(dataFile, 0L, SEEK_END);
    nFloats = ftell(dataFile) / 4;
    fseek(dataFile, 0L, SEEK_SET);

    // get remaining memory on card
    size_t deviceFree, deviceTotal;
    hipMemGetInfo(&deviceFree, &deviceTotal);

    size_t chunkSize;
    size_t inputFrameSize = d->layerSize[0] * 16;

    // if the dataset fits in 50% of remaining memory, allocate the whole she-bang
    if( nFloats < (size_t) ((float) (deviceFree/4) * 0.8) )
    {
        chunkSize = nFloats;
    }
    else
    {
        // otherwise, use ~50% of remaining memory for data chunks (rounding up to the next digit presentation size)
        chunkSize = (size_t) ((float) (deviceFree/4) * 0.8 );
        chunkSize += inputFrameSize*d->nMovements - (chunkSize % (inputFrameSize*d->nMovements));
    }

    // allocate space for data set
    MALLOC(d->dataSet, float, chunkSize);
    CUDAMALLOC( (void **) &d->dataSet_dev, sizeof(float)*chunkSize );

    size_t nFloatsRead = 0;

    int i, iMod;

    // while the whole file hasn't been read...
    while( nFloatsRead < nFloats )
    {
        size_t nFloats_it = 0;

        // read in a chunk
        i = 0;
        while( i < chunkSize )
        {
            if( feof( dataFile ) )
            {
                break;
            }
            nFloats_it += fread( &d->dataSet[i], sizeof(float), inputFrameSize, dataFile );

            i += inputFrameSize;
        }

        nFloatsRead += nFloats_it;

        // copy chunk of data set to device
        CUDAMEMCPY( d->dataSet_dev, d->dataSet, sizeof(float) * chunkSize, hipMemcpyHostToDevice );

        // get number of iterations to run
        int nIt = nFloats_it / inputFrameSize;

        printf("presenting %d movements...\n", nIt);

        for( i=0; i < nIt; i++ )
        {
            // a new digit is picked up every d->nMovements movements.
            if( i % d->nMovements == 0 )
            {
                ClearBeliefs( d );
            }

            // formulate belief/update on the presentation
            FormulateBelief( d, isTrain, d->dataSet_dev + i*inputFrameSize );

            // write out beliefs if we aren't training
            if( !isTrain )
            {
                // write out the 10,12,14 movements
                iMod = i % d->nMovements;
                if( iMod == 10 || iMod == 12 || iMod == 14 )
                {
                    CUDAMEMCPY( d->belief, d->belief_dev, sizeof(float) * d->nBeliefs, hipMemcpyDeviceToHost );
                    fwrite( d->belief, sizeof(float), d->nBeliefs, beliefFile );
                }
            }
        }
    }

    fclose( dataFile );

    if( !isTrain )
    {
        fclose( beliefFile );
    }

    FREE( d->dataSet );
    CUDAFREE( d->dataSet_dev );
}

void CopyDestinToDevice( Destin *d )
{
    int i;

    // copy individual nodes
    for( i=0; i < d->nNodes; i++ )
    {
        CopyNodeToDevice( &d->nodes_host[i] );
    }

    // copy belief and input information
    CUDAMEMCPY( d->belief_dev,        d->belief,        sizeof(float) * d->nBeliefs,       hipMemcpyHostToDevice );
    CUDAMEMCPY( d->inputPipeline_dev, d->inputPipeline, sizeof(float) * d->nInputPipeline, hipMemcpyHostToDevice );
}

void CopyDestinFromDevice( Destin *d )
{
    int i;

    // copy individual nodes
    for( i=0; i < d->nNodes; i++)
    {
        CopyNodeFromDevice( &d->nodes_host[i] );
    }

    // copy belief and input information
    CUDAMEMCPY( d->belief,        d->belief_dev,        sizeof(float) * d->nBeliefs,       hipMemcpyDeviceToHost );
    CUDAMEMCPY( d->inputPipeline, d->inputPipeline_dev, sizeof(float) * d->nInputPipeline, hipMemcpyDeviceToHost );
}

void DestroyDestin( Destin *d )
{
    int i;

    for( i=0; i < d->nNodes; i++ )
    {
        DestroyNode( &d->nodes_host[i] );
    }

    FREE(d->nodes_host);
    FREE(d->inputPipeline);
    FREE(d->belief);

    FREE(d->layerSize);

    CUDAFREE(d->nodes_dev);
    CUDAFREE(d->inputPipeline_dev);
    CUDAFREE(d->belief_dev);

    FREE(d);
}

// present the network with an image pointed to by image_dev.  update the network if doUpdate is true,
// otherwise just change the belief.
void FormulateBelief(Destin *d, bool isTrain, float *image_dev)
{
    dim3 blockSize(d->nNodes, d->maxNb);

    CalculateDistances<<<blockSize,d->maxNs,sizeof(float)*d->maxNs*2>>>(d->nodes_dev, image_dev);

    if( isTrain )
    // normalize belief and update winner
    {
        NormalizeBeliefGetWinner<<<d->nNodes,d->maxNb,sizeof(float)*d->maxNb*4>>>(d->nodes_dev);
        UpdateWinner<<<d->nNodes,d->maxNs>>>(d->nodes_dev, image_dev);
    }
    else
    // just normalize the belief
    {
        NormalizeBelief<<<d->nNodes,d->maxNb,sizeof(float)*d->maxNb*2>>>(d->nodes_dev);
    }

    CUDAMEMCPY( d->inputPipeline_dev, d->belief_dev, sizeof(float)*d->nInputPipeline, hipMemcpyDeviceToDevice );

    CUDACHECKERROR();

    return;
}

// zeros-out the beliefs (and input registers) for each node.  useful for "restarting" the network
// between presentations of distinct elements
void ClearBeliefs( Destin *d )
{
    int i;

    for( i=0; i < d->nInputPipeline; i++ )
    {
        d->inputPipeline[i] = 0;
    }

    for( i=0; i < d->nBeliefs; i++ )
    {
        d->belief[i] = 0;
    }
    
    // copy to card
    CUDAMEMCPY( d->inputPipeline_dev, d->inputPipeline, sizeof(float) * d->nInputPipeline, hipMemcpyHostToDevice);
    CUDAMEMCPY( d->belief_dev,        d->belief,        sizeof(float) * d->nBeliefs,       hipMemcpyHostToDevice);
}
