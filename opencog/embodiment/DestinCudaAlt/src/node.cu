#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>

#include "node.h"
#include "macros.h"

#include "hip/device_functions.h"

#define ALPHA       0.01
#define BETA        0.01
#define LAMBDA      0.5
#define GAMMA       1
#define STARVCOEFF  0.05
#define EPSILON     1e-25
#define LARGE_FLOAT 10

void PrintNode( Node *n )
{
    CudaNode cn;

    printf("node statistics\n");
    printf("nb: 0x%x    ni: 0x%x    np: 0x%x    ns: 0x%x    nb*ns: 0x%x\n", n->nb, n->ni, n->np, n->ns, n->nb*n->ns);

/*
    printf("Host pointers:\n");
    printf("mu: %p\n", n->mu);
    printf("sigma: %p\n", n->sigma);
    printf("starv: %p\n", n->starv);
    printf("input: %p\n", n->input);
    printf("inputOffsets: %p\n", n->inputOffsets);
    printf("observation: %p\n", n->mu);
    printf("beliefEuc: %p\n", n->beliefEuc);
    printf("beliefMal: %p\n", n->beliefMal);
    printf("pBelief: %p\n", n->pBelief);
    printf("parent_pBelief: %p\n\n", n->parent_pBelief);
*/

    uint nb = n->nb;
    uint ni = n->ni;
    uint np = n->np;
    uint ns = n->ns;

    uint muSize = n->nb * n->ns;

    CUDAMEMCPY( &cn, n->node_dev, sizeof(CudaNode), hipMemcpyDeviceToHost );

    printf("Device pointers:\n");
    printf("mu: %p to %p\n", cn.mu, cn.mu + muSize);
    printf("sigma: %p to %p\n", cn.sigma, &cn.sigma[muSize]);
    printf("starv: %p to %p\n", cn.starv, &cn.starv[nb] );
    printf("beliefEuc: %p to %p\n", cn.beliefEuc, &cn.beliefEuc[nb]);
    printf("beliefMal: %p to %p\n", cn.beliefMal, &cn.beliefMal[nb]);
    printf("observation: %p to %p\n", cn.observation, &cn.observation[ns] );

    printf("input: %p to %p\n", cn.input, &cn.input[ni] );
    printf("inputOffsets: %p to %p\n", cn.inputOffsets, &cn.inputOffsets[ni] );

    printf("pBelief: %p to %p\n", cn.pBelief, &cn.pBelief[nb] );
    printf("parent_pBelief: %p to %p\n", cn.parent_pBelief, &cn.parent_pBelief[np]);
}

uint NodeStatsSize(int ni, int nb, int np)
{
    uint ns = ni + nb + np;
    return nb * ns * 2 + nb * 3 + ns;
}

// Initialize a node
void InitNode
    (
    uint         nodeIdx,
    uint         ni,
    uint         nb,
    uint         np,
    float       starvCoeff,
    float       alpha,
    float       beta,
    Node        *node_host,
    CudaNode    *cudaNode_dev,
    uint        *inputOffsets,
    uint        *inputOffsetMemory_dev,
    float       *input_dev,
    float       *input_host,
    float       *belief_dev,
    float       *belief_host,
    float       *statsMemory_dev

    )
{

    CudaNode cudaNode_host;
    
    // calculate the state dimensionality (number of inputs + number of beliefs)
    uint ns = ni+nb+np;

    if( ns > THREADS_MAX )
    {
        fprintf(stderr, "State dimensionality is too high.\n");
        fprintf(stderr, "Node index: %d.  ni: %d. nb: %d. np: %d. ns: %d\n", nodeIdx, ni, nb, np, ns);
        exit(1);
    }

    // link host with device pointer
    node_host->node_dev = cudaNode_dev;

    // Initialize node parameters
    node_host->nb            = nb;
    node_host->ni            = ni;
    node_host->np            = np;
    node_host->ns            = ns;
    node_host->starvCoeff    = starvCoeff;
    node_host->alpha         = alpha;
    node_host->beta          = beta;
    node_host->winner        = 0;

    // allocate space on host
    MALLOC( node_host->memory_area , float , (nb * ns * 2 + nb * 3 + ns) );

    //use pointer arithmetic to divide the memory
    node_host->mu =          node_host->memory_area;
    node_host->sigma =       node_host->mu         + nb*ns;
    node_host->starv =       node_host->sigma      + nb*ns;
    node_host->beliefEuc =   node_host->starv      + nb;
    node_host->beliefMal =   node_host->beliefEuc  + nb;
    node_host->observation = node_host->beliefMal  + nb;

    // point to the block-allocated space
    node_host->input = input_host;
    node_host->pBelief = belief_host;

    // Initialize node parameters
    cudaNode_host.nb            = nb;
    cudaNode_host.ni            = ni;
    cudaNode_host.np            = np;
    cudaNode_host.ns            = ns;
    cudaNode_host.starvCoeff    = starvCoeff;
    cudaNode_host.alpha         = alpha;
    cudaNode_host.beta          = beta;
    cudaNode_host.winner        = 0;

    // allocate node statistics on device using pointer arithmetic to divide up the memory
    cudaNode_host.mu =          statsMemory_dev;
    cudaNode_host.sigma =       cudaNode_host.mu        + nb*ns;
    cudaNode_host.starv =       cudaNode_host.sigma     + nb*ns;
    cudaNode_host.beliefEuc =   cudaNode_host.starv     + nb;
    cudaNode_host.beliefMal =   cudaNode_host.beliefEuc + nb;
    cudaNode_host.observation = cudaNode_host.beliefMal + nb;

    // point to the block-allocated space 
    cudaNode_host.input = input_dev;
    cudaNode_host.pBelief = belief_dev;

    // copy the input offset for the inputs (should be NULL for non-input nodes)
    if( inputOffsets != NULL )
    {
        MALLOC(node_host->inputOffsets, uint, ni);
        memcpy(node_host->inputOffsets, inputOffsets, sizeof(uint) * ni);

        cudaNode_host.inputOffsets = inputOffsetMemory_dev;
        CUDAMEMCPY( cudaNode_host.inputOffsets, node_host->inputOffsets, sizeof(uint) * ni, hipMemcpyHostToDevice);
    }
    else
    {
        node_host->inputOffsets = NULL;
        cudaNode_host.inputOffsets = NULL;
    }

    uint i,j;
    for( i=0; i < nb; i++ )
    {
        // init belief (node output)
        node_host->pBelief[i] = 1 / nb;
        node_host->beliefEuc[i] = 1 / nb;
        node_host->beliefMal[i] = 1 / nb;

        // init starv trace to one
        node_host->starv[i] = 1.0f;

        // init mu and sigma
        for(j=0; j < ns; j++)
        {
            node_host->mu[i*ns+j] = (float) rand() / (float) RAND_MAX;
            node_host->sigma[i*ns+j] = 0.00001;
        }
    }

    for( i=0; i < ns; i++ )
    {
        node_host->observation[i] = 0;
    }

    // copy initialized statistics to device
    CUDAMEMCPY( cudaNode_host.mu,           node_host->mu,          sizeof(float) * nb * ns,    hipMemcpyHostToDevice );
    CUDAMEMCPY( cudaNode_host.sigma,        node_host->sigma,       sizeof(float) * nb * ns,    hipMemcpyHostToDevice );
    CUDAMEMCPY( cudaNode_host.starv,        node_host->starv,       sizeof(float) * nb,         hipMemcpyHostToDevice );
    CUDAMEMCPY( cudaNode_host.pBelief,      node_host->pBelief,     sizeof(float) * nb,         hipMemcpyHostToDevice );
    CUDAMEMCPY( cudaNode_host.observation,  node_host->observation, sizeof(float) * ns,         hipMemcpyHostToDevice );

    // copy the node struct to the device
    CUDAMEMCPY( cudaNode_dev,           &cudaNode_host,     sizeof(CudaNode),           hipMemcpyHostToDevice );
}

// deallocate the node.
void DestroyNode( Node *n )
{
    // free host data
    // free host memory for mu, sigma, starv, pBelief, beliefEuc, beliefMal
    FREE(n->memory_area);

    // if it is a zero-layer node, free the input offset array on the host
    if( n->inputOffsets != NULL)
    {
        FREE(n->inputOffsets);
    }

    // free device data
    CudaNode cudaNode_host;
    CUDAMEMCPY( &cudaNode_host, n->node_dev, sizeof(CudaNode), hipMemcpyDeviceToHost );
}

// copy the node statistics from the host to the device.
void CopyNodeToDevice(Node *host)
{
    CudaNode cudaNode_host;

    // copy struct from device
    CUDAMEMCPY( &cudaNode_host, host->node_dev, sizeof(CudaNode), hipMemcpyDeviceToHost );

    cudaNode_host.winner = host->winner;

    // copy to pointers given from the struct
    CUDAMEMCPY( cudaNode_host.mu,           host->mu,           sizeof(float)*host->nb*host->ns,    hipMemcpyHostToDevice );
    CUDAMEMCPY( cudaNode_host.sigma,        host->sigma,        sizeof(float)*host->nb*host->ns,    hipMemcpyHostToDevice );
    CUDAMEMCPY( cudaNode_host.starv,        host->starv,        sizeof(float)*host->nb,             hipMemcpyHostToDevice );
    CUDAMEMCPY( cudaNode_host.pBelief,      host->pBelief,      sizeof(float)*host->nb,             hipMemcpyHostToDevice );
    CUDAMEMCPY( cudaNode_host.beliefEuc,    host->beliefEuc,    sizeof(float)*host->nb,             hipMemcpyHostToDevice );
    CUDAMEMCPY( cudaNode_host.beliefMal,    host->beliefMal,    sizeof(float)*host->nb,             hipMemcpyHostToDevice );
    CUDAMEMCPY( cudaNode_host.observation,  host->observation,  sizeof(float)*host->ns,             hipMemcpyHostToDevice );

    CUDAMEMCPY( host->node_dev,          &cudaNode_host,     sizeof(CudaNode),                   hipMemcpyHostToDevice );
}

// copy the node statistics from the device to the host.
void CopyNodeFromDevice(Node *host)
{
    CudaNode cudaNode_host;

    // copy struct from device
    CUDAMEMCPY( &cudaNode_host, host->node_dev, sizeof(CudaNode), hipMemcpyDeviceToHost );

    host->winner = cudaNode_host.winner;

    if( cudaNode_host.inputOffsets == NULL )
    {
        CUDAMEMCPY( host->input, cudaNode_host.input, sizeof(float)*host->ni, hipMemcpyDeviceToHost );
    }

    // copy from pointers given from the struct
    CUDAMEMCPY( host->mu,           cudaNode_host.mu,           sizeof(float)*host->nb*host->ns,    hipMemcpyDeviceToHost );
    CUDAMEMCPY( host->sigma,        cudaNode_host.sigma,        sizeof(float)*host->nb*host->ns,    hipMemcpyDeviceToHost );
    CUDAMEMCPY( host->starv,        cudaNode_host.starv,        sizeof(float)*host->nb,             hipMemcpyDeviceToHost );
    CUDAMEMCPY( host->pBelief,      cudaNode_host.pBelief,      sizeof(float)*host->nb,             hipMemcpyDeviceToHost );
    CUDAMEMCPY( host->beliefEuc,    cudaNode_host.beliefEuc,    sizeof(float)*host->nb,             hipMemcpyDeviceToHost );
    CUDAMEMCPY( host->beliefMal,    cudaNode_host.beliefMal,    sizeof(float)*host->nb,             hipMemcpyDeviceToHost );
    CUDAMEMCPY( host->observation,  cudaNode_host.observation,  sizeof(float)*host->ns,             hipMemcpyDeviceToHost );
}

// GetObservation:
//      CUDA kernel that creates the observation for a node
//      given its input (input image or a previous node's belief),
//      previous belief, and parent's belief.
__global__ void GetObservation( CudaNode *n, float *framePtr )
{
    n = &n[blockIdx.x];

    uint ns = n->ns;
    uint ni = n->ni;
    uint nb = n->nb;
    uint np = n->np;

    if( threadIdx.x < ns )
    {
        if( threadIdx.x < ni )
        {
            if( n->inputOffsets == NULL )
            {
                n->observation[threadIdx.x] = n->input[threadIdx.x];
            }
            else
            {
                n->observation[threadIdx.x] = framePtr[n->inputOffsets[threadIdx.x]];
            }
        }
        else if( threadIdx.x < ni + nb )
        {
            n->observation[threadIdx.x] = n->pBelief[threadIdx.x-ni];
        }
        else
        {
            if( np > 0 )
            {
                n->observation[threadIdx.x] = n->parent_pBelief[threadIdx.x-ni-nb];
            }
        }
    }
}

// CPU implementation of GetObservation kernel
void __CPU_GetObservation( Node *n, float *framePtr, uint nIdx )
{
    n = &n[nIdx];

    uint i;

    for( i=0; i < n->ni; i++ )
    {
        n->observation[i] = (n->inputOffsets == NULL) ? n->input[i] : framePtr[n->inputOffsets[i]];
    }

    for( i=n->ni; i < n->ni+n->nb; i++ )
    {
        n->observation[i] = n->pBelief[i-n->ni];
    }

    for( i=n->ni+n->nb; i < n->ns; i++ )
    {
        n->observation[i] = n->parent_pBelief[i-n->ni-n->nb];
    }
}

//  CalculateDistances:
//      CUDA kernel that takes in a node and an observation
//        and writes the unnormalized belief to the particular
//        node.
//
//
//              block = (number of nodes) x (max belief dimensionality)
//             thread = (max state dimensionality) x 1
//         shared mem = euclidean sum  (length: ns)
//                      malhanobis sum (length: ns)
__global__ void CalculateDistances( CudaNode *n )
{
    // shared array -- includes the euclidean and malhanobis arrays for reduction
    extern __shared__ float shared[];
    
    // value for how far the observation (or collection of beliefs from a set of previous
    // nodes) deviates from each centroid
    float delta;

    // pointers in shared memory to the euclidean and malhanobis sums
    float *sumEuc, *sumMal;

    // grab pointer to the node we want to get distances for
    n = &n[blockIdx.x];
    uint i;          // iterator for the reduction
    uint mIdx;       // entry in the mu/sigma matrix to calculate delta
    uint ns;         // node state size

    ns = n->ns;

    // maxNS is likely greater than n->ns.  don't execute if we're not in range.
    // we do this because the state dimensionalities differ between nodes, but
    // we can't modify the size of the kernel that is called.  if the state
    // dimensionality is out of range, effectively execute a no-op.
    if( blockIdx.y < n->nb && threadIdx.x < ns )
    {
        // point euclidean and malhanobis arrays to shared mem
        sumEuc = (float *) &shared[ns*0];
        sumMal = (float *) &shared[ns*1];

        // get entry in the mu/sigma matrices to calculate for this thread
        mIdx = blockIdx.y * ns + threadIdx.x;

        // get difference for each dimension between the input state (input + prev belief) and every
        // centroid location
        delta = n->mu[mIdx] - n->observation[threadIdx.x];

        delta *= n->starv[blockIdx.y];
        delta *= delta;

        sumEuc[threadIdx.x] = delta;
        sumMal[threadIdx.x] = (n->sigma[mIdx] < EPSILON) ? LARGE_FLOAT  : (delta / n->sigma[mIdx]);

        // sync threads before summing up the columns
        __syncthreads();

        // reduce euc and mal partial sums.  total sum will be in
        // sumEuc[0] and sumMal[0].
        // ** THIS BLOCK NEEDS WORK **
        // It is awful for two reasons -- it is highly divergent,
        // and it causes a huge number of bank conflicts.
        for( i=1; i < ns; i <<= 1 )
        {
            if( threadIdx.x % (i*2) == 0 && threadIdx.x + i < ns )
            {
        /*
        for( i = ns / 2; i > 0; i >>= 1 )
        {
            if( threadIdx.x < i && threadIdx.x+i < ns )
            {
        */
                sumEuc[threadIdx.x] += sumEuc[threadIdx.x+i];
                sumMal[threadIdx.x] += sumMal[threadIdx.x+i];
            }
            __syncthreads();
        }
        
        // get inverse of distance (provides "confidence" or a value of
        // closeness from the centroid to the observation)
        if( threadIdx.x == 0 )
        {
            sumEuc[0] = sqrt(sumEuc[0]);
            sumMal[0] = sqrt(sumMal[0]);

            n->beliefEuc[blockIdx.y] = (sumEuc[0] < EPSILON) ? 1 : (1 / sumEuc[0]);
            n->beliefMal[blockIdx.y] = (sumMal[0] < EPSILON) ? 1 : (1 / sumMal[0]);
        }
    }
}

// CPU implementation of CalculateDistances kernel
void __CPU_CalculateDistances( Node *n, uint nIdx )
{
    float delta;
    float sumEuc, sumMal;

    n = &n[nIdx];

    uint i, j;

    // iterate over each belief
    for( i=0; i < n->nb; i++ )
    {
        sumEuc = 0;
        sumMal = 0;

        // iterate over each state for belief
        for( j=0; j < n->ns; j++ )
        {
            delta = n->mu[i*n->ns+j] - n->observation[j];

            sumEuc += delta * delta * n->starv[i];
            sumMal += delta * delta * n->starv[i] / n->sigma[i*n->ns+j];
        }

        sumEuc = sqrt(sumEuc);
        sumMal = sqrt(sumMal);

        n->beliefEuc[i] = ( sumEuc < EPSILON ) ? 1 : (1 / sumEuc);
        n->beliefMal[i] = ( sumMal < EPSILON ) ? 1 : (1 / sumMal);
    }
}

// NormalizeBelief:
//      CUDA kernel that normalizes the belief of a node such that all
//      the individual components sum to 1.  This gives each belief a
//      probability that the observation is "close" to a centroid.
//
//                block = individual node
//               thread = belief dimension for a node
//           shared mem = euclidean sum
//                        malhanobis sum

__global__ void NormalizeBelief(CudaNode *n)
{
    extern __shared__ float shared[];
    uint i;

    // grab the node we want
    n = &n[blockIdx.x];

    uint nb;         // number of centroids

    nb = n->nb;

    if( threadIdx.x < nb )
    {
        float *normEuc = (float *) &shared[nb*0];
        float *normMal = (float *) &shared[nb*1];

        // copy normEuc and normMal from global memory
        normEuc[threadIdx.x] = n->beliefEuc[threadIdx.x];
        normMal[threadIdx.x] = n->beliefMal[threadIdx.x];

        // make sure normEuc and normMal are completely populated
        __syncthreads();

        // calculate the normalization constant for the belief
        // this reduction is slow and awful, see above
        for( i=1; i < nb; i <<= 1 )
        {
            if( threadIdx.x % (i*2) == 0 && threadIdx.x + i < nb )
            {
        /*
        for( i = nb / 2; i > 0; i >>= 1 )
        {
            if( threadIdx.x < i && threadIdx.x+i < nb )
            {
        */
                normEuc[threadIdx.x] += normEuc[threadIdx.x + i];
                normMal[threadIdx.x] += normMal[threadIdx.x + i];
            }

            __syncthreads();
        }

        n->beliefEuc[threadIdx.x] = ( normEuc[0] < EPSILON ) ? (1 / (float) n->nb) : (n->beliefEuc[threadIdx.x] / normEuc[0]);
        n->beliefMal[threadIdx.x] = ( normMal[0] < EPSILON ) ? (1 / (float) n->nb) : (n->beliefMal[threadIdx.x] / normMal[0]);

        // update belief
        n->pBelief[threadIdx.x] = n->beliefMal[threadIdx.x];
    }
}

// CPU implementation of NormalizeBelief kernel
void __CPU_NormalizeBelief( Node *n, uint nIdx )
{
    n = &n[nIdx];
    
    float normEuc = 0;
    float normMal = 0;

    uint i;

    for( i=0; i < n->nb; i++ )
    {
        normEuc += n->beliefEuc[i];
        normMal += n->beliefMal[i];
    }

    for( i=0; i < n->nb; i++ )
    {
        n->beliefEuc[i] = ( normEuc < EPSILON ) ? (1 / (float) n->nb) : (n->beliefEuc[i] / normEuc);
        n->beliefMal[i] = ( normMal < EPSILON ) ? (1 / (float) n->nb) : (n->beliefMal[i] / normMal);
        n->pBelief[i] = n->beliefMal[i];
    }

}


// NormalizeBeliefGetWinner:
//      CUDA kernel that normalizes the belief (as above) and determines the winning
//        centroid
//
//                block = individual node
//               thread = belief dimension for a node
//           shared mem = euclidean sum
//                        max euclidean belief (to determine the winning centroid)
//                        malhanobis sum
//                        max euclidean belief index (to modify the winning centroid)

__global__ void NormalizeBeliefGetWinner( CudaNode *n )
{
    extern __shared__ float shared[];
    
    uint     i;

    float  *normEuc, *maxEuc, *normMal;
    uint    *maxIdx;

    // pick our particular node
    n = &n[blockIdx.x];

    uint nb;         // number of centroids

    nb = n->nb;

    if( threadIdx.x < nb )
    {
        // set up shared pointers
        normEuc = (float *) &shared[nb*0];
        normMal = (float *) &shared[nb*1];
        maxEuc = (float *) &shared[nb*2];
        maxIdx = (uint *) &shared[nb*3];

        // populate shared memory for reductions
        normEuc[threadIdx.x] = n->beliefEuc[threadIdx.x];
        normMal[threadIdx.x] = n->beliefMal[threadIdx.x];
        maxEuc[threadIdx.x] = n->beliefEuc[threadIdx.x];
        maxIdx[threadIdx.x] = threadIdx.x;

        __syncthreads();

        // this reduction is slow and awful, see above
        for( i=1; i < nb; i <<= 1 )
        {
            if( threadIdx.x % (i*2) == 0 && threadIdx.x + i < nb )
            {
        /*
        for( i = nb / 2; i > 0; i >>= 1 )
        {
            if( threadIdx.x < i && threadIdx.x+i < nb )
            {
        */
                // reduce euclidean and malhanobis sums
                normEuc[threadIdx.x] += normEuc[threadIdx.x + i];
                normMal[threadIdx.x] += normMal[threadIdx.x + i];

                // reduce winning centroid
                if( maxEuc[threadIdx.x] < maxEuc[threadIdx.x + i] )
                {
                    maxEuc[threadIdx.x] = maxEuc[threadIdx.x + i];
                    maxIdx[threadIdx.x] = maxIdx[threadIdx.x + i];
                } 
            }

            __syncthreads();
        }
        
        n->beliefEuc[threadIdx.x] = ( normEuc[0] < EPSILON ) ? (1 / n->nb) : (n->beliefEuc[threadIdx.x] / normEuc[0]);
        n->beliefMal[threadIdx.x] = ( normMal[0] < EPSILON ) ? (1 / n->nb) : (n->beliefMal[threadIdx.x] / normMal[0]);

        if( threadIdx.x == 0 )
        {
            n->winner = maxIdx[0];
        }
    }
}


// CPU implementation of NormalizeBelief kernel
void __CPU_NormalizeBeliefGetWinner( Node *n, uint nIdx )
{
    n = &n[nIdx];
    
    float normEuc = 0;
    float normMal = 0;

    float maxEucVal = n->beliefEuc[0];
    uint maxEucIdx = 0;

    uint i;

    for( i=0; i < n->nb; i++ )
    {
        normEuc += n->beliefEuc[i];
        normMal += n->beliefMal[i];

        if( n->beliefEuc[i] > maxEucVal )
        {
            maxEucVal = n->beliefEuc[i];
            maxEucIdx = i;
        }
    }

    for( i=0; i < n->nb; i++ )
    {
        n->beliefEuc[i] = ( normEuc < EPSILON ) ? (1 / n->nb) : (n->beliefEuc[i] / normEuc);
        n->beliefMal[i] = ( normMal < EPSILON ) ? (1 / n->nb) : (n->beliefMal[i] / normMal);
    }

    n->winner = maxEucIdx;
}

//  UpdateWinner:
//    CUDA Kernel that updates the winning centroid
//
//              block = (number of nodes) x 1
//             thread = (max state dimensionality) x 1
__global__ void UpdateWinner( CudaNode *n )
{
    // value for how far the observation (or collection of beliefs from a set of previous
    // nodes) deviates from each centroid
    float delta;

    // grab pointer to the node we want to get distances for
    n = &n[blockIdx.x];

    uint mIdx;       // entry in the mu/sigma matrix to calculate delta

    uint nb;         // number of centroids
    uint ns;         // state dimensionality
    uint winner;     // winner idx

    nb = n->nb;
    ns = n->ns;
    winner = n->winner;

    // maxNS is likely greater than ns.  don't execute if we're not in range.
    // we do this because the state dimensionalities differ between nodes, but
    // we can't modify the size of the kernel that is called.  if the state
    // dimensionality is out of range, effectively execute a no-op.
    if( threadIdx.x < ns )
    {
        // get entry in the mu/sigma matrices to calculate for this thread
        mIdx = winner * ns + threadIdx.x;
        
        // get difference for each dimension between the input state (input + prev belief) and every
        // centroid location
        delta = n->mu[mIdx] - n->observation[threadIdx.x];

        // update mu and sigma
        n->mu[mIdx] -= ALPHA*delta;
        n->sigma[mIdx] -= BETA*(n->sigma[mIdx] - delta*delta);
    }

    // update starvation and belief
    if( threadIdx.x < nb )
    {
        n->starv[threadIdx.x] *= 1 - STARVCOEFF;

        if( threadIdx.x == n->winner )
        {
            n->starv[threadIdx.x] += STARVCOEFF;
        }

        n->pBelief[threadIdx.x] = n->beliefMal[threadIdx.x];
    }
}

// CPU implementation for UpdateWinner kernel
void __CPU_UpdateWinner( Node *n, uint nIdx )
{
    n = &n[nIdx];

    uint i;
    uint winnerOffset = n->winner*n->ns;
    float delta;

    for( i=0; i < n->ns; i++ )
    {
        delta = n->mu[winnerOffset+i] - n->observation[i];

        n->mu[winnerOffset+i] -= ALPHA * delta;
        n->sigma[winnerOffset+i] -= BETA * (n->sigma[winnerOffset+i] - delta*delta);
    }

    for( i=0; i < n->nb; i++ )
    {
        n->starv[i] *= 1 - STARVCOEFF;
        n->pBelief[i] = n->beliefMal[i];
    }

    n->starv[n->winner] += STARVCOEFF;
}

// a quick function to print remaining memory on the card (helps to debug any
// memory leak issues)
void cudaPrintMemory()
{
    size_t mfree, mtotal;

    hipMemGetInfo(&mfree, &mtotal);
    printf("Usage: %012zu/%012zu\n", mfree, mtotal);
}
