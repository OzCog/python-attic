#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>

#include "node.h"
#include "macros.h"

#include "hip/device_functions.h"

#define ALPHA       0.01
#define BETA        0.01
#define LAMBDA      0.5
#define GAMMA       1
#define STARVCOEFF  0.05


uint NodeStatsSize(int ni, int nb, int np)
{
    uint ns = ni + nb + np;
    return nb * ns * 2 + nb * 4;
}

// Initialize a node
void InitNode
    (
    uint         nodeIdx,
    uint         ni,
    uint         nb,
    uint         np,
    float       starvCoeff,
    float       alpha,
    float       beta,
    Node        *node_host,
    CudaNode    *cudaNode_dev,
    uint        *inputOffsets,
    uint        *inputOffsetMemory_dev,
    float       *input_dev,
    float       *belief_dev,
    float       *statsMemory_dev

    )
{

    CudaNode cudaNode_host;
    
    // calculate the state dimensionality (number of inputs + number of beliefs)
    uint ns = ni+nb+np;

    if( ns > THREADS_MAX )
    {
        fprintf(stderr, "State dimensionality is too high.\n");
        fprintf(stderr, "Node index: %d.  ni: %d. nb: %d. np: %d. ns: %d\n", nodeIdx, ni, nb, np, ns);
        exit(1);
    }

    // link host with device pointer
    node_host->node_dev = cudaNode_dev;

    // Initialize node parameters
    node_host->nb            = nb;
    node_host->ni            = ni;
    node_host->ns            = ns;
    node_host->np            = np;
    node_host->starvCoeff    = starvCoeff;
    node_host->alpha         = alpha;
    node_host->beta          = beta;
    node_host->winner        = 0;


    // allocate space on host
    MALLOC( node_host->memory_area , float , (nb * ns * 2 + nb * 4) );
    //use pointer arithmetic to divide the memory
    node_host->mu =         node_host->memory_area;
    node_host->sigma =      node_host->mu         + nb*ns;
    node_host->starv =      node_host->sigma      + nb*ns;
    node_host->pBelief =    node_host->starv      + nb;
    node_host->beliefEuc =  node_host->pBelief    + nb;
    node_host->beliefMal =  node_host->beliefEuc  + nb;

    if( inputOffsets == NULL )
    {
        MALLOC(node_host->input, float, ni);
    }
    else
    {
        node_host->input = NULL;
    }

    // Initialize node parameters
    cudaNode_host.nb            = nb;
    cudaNode_host.ni            = ni;
    cudaNode_host.np            = np;
    cudaNode_host.ns            = ns;
    cudaNode_host.starvCoeff    = starvCoeff;
    cudaNode_host.alpha         = alpha;
    cudaNode_host.beta          = beta;

    // allocate node statistics on device using pointer arithmetic to divide up the memory
    cudaNode_host.mu =          statsMemory_dev;
    cudaNode_host.sigma =       cudaNode_host.mu        + nb*ns;
    cudaNode_host.starv =       cudaNode_host.sigma     + nb*ns;
    cudaNode_host.beliefEuc =   cudaNode_host.starv     + nb;
    cudaNode_host.beliefMal =   cudaNode_host.beliefEuc + nb;
    cudaNode_host.dist =        cudaNode_host.beliefMal + nb;

    // point to the space allocated for the input (should be NULL for input nodes)
    cudaNode_host.input = input_dev;

    // copy the input offset for the inputs (should be NULL for non-input nodes)
    if( inputOffsets != NULL )
    {
        MALLOC(node_host->inputOffsets, uint, ni);
        memcpy(node_host->inputOffsets, inputOffsets, sizeof(uint) * ni);
        cudaNode_host.inputOffsets = inputOffsetMemory_dev;
        CUDAMEMCPY( cudaNode_host.inputOffsets, node_host->inputOffsets, sizeof(uint) * ni, hipMemcpyHostToDevice);
    }
    else
    {
        node_host->inputOffsets = NULL;
        cudaNode_host.inputOffsets = NULL;
    }


    // set prior belief to block-allocated value
    cudaNode_host.pBelief = belief_dev;

    uint i,j;

    for(i=0; i < nb; i++)
    {
        // init belief (node output)
        node_host->pBelief[i] = 0;

        // init starv trace to one
        node_host->starv[i] = 1.0f;

        // init mu and sigma
        for(j=0; j < ns; j++)
        {
            node_host->mu[i*ns+j] = (float) rand() / (float) RAND_MAX;
            node_host->sigma[i*ns+j] = 0.00001;
        }
    }

    // copy initialized statistics to device
    CUDAMEMCPY( cudaNode_host.mu,       node_host->mu,      sizeof(float) * nb * ns,    hipMemcpyHostToDevice );
    CUDAMEMCPY( cudaNode_host.sigma,    node_host->sigma,   sizeof(float) * nb * ns,    hipMemcpyHostToDevice );
    CUDAMEMCPY( cudaNode_host.starv,    node_host->starv,   sizeof(float) * nb,         hipMemcpyHostToDevice );
    CUDAMEMCPY( cudaNode_host.pBelief,  node_host->pBelief, sizeof(float) * nb,         hipMemcpyHostToDevice );

    // copy the node struct to the device
    CUDAMEMCPY( cudaNode_dev,           &cudaNode_host,     sizeof(CudaNode),           hipMemcpyHostToDevice );
}

// deallocate the node.
void DestroyNode( Node *n )
{
    // free host data
    // free host memory for mu, sigma, starv, pBelief, beliefEuc, beliefMal
    FREE(n->memory_area);

    // if it is a zero-layer node, free the input offset array on the host
    if( n->inputOffsets != NULL)
    {
        FREE(n->inputOffsets);
    }
    else
    {
        FREE(n->input);
    }

    // free device data
    CudaNode cudaNode_host;
    CUDAMEMCPY( &cudaNode_host, n->node_dev, sizeof(CudaNode), hipMemcpyDeviceToHost );

}

// copy the node statistics from the host to the device.
void CopyNodeToDevice(Node *host)
{
    CudaNode cudaNode_host;

    // copy struct from device
    CUDAMEMCPY( &cudaNode_host, host->node_dev, sizeof(CudaNode), hipMemcpyDeviceToHost );

    cudaNode_host.winner = host->winner;

    // copy to pointers given from the struct
    CUDAMEMCPY( cudaNode_host.mu,        host->mu,           sizeof(float)*host->nb*host->ns,    hipMemcpyHostToDevice );
    CUDAMEMCPY( cudaNode_host.sigma,     host->sigma,        sizeof(float)*host->nb*host->ns,    hipMemcpyHostToDevice );
    CUDAMEMCPY( cudaNode_host.starv,     host->starv,        sizeof(float)*host->nb,             hipMemcpyHostToDevice );
    CUDAMEMCPY( cudaNode_host.beliefEuc, host->beliefEuc,    sizeof(float)*host->nb,             hipMemcpyHostToDevice );
    CUDAMEMCPY( cudaNode_host.beliefMal, host->beliefMal,    sizeof(float)*host->nb,             hipMemcpyHostToDevice );
    CUDAMEMCPY( cudaNode_host.pBelief,   host->pBelief,      sizeof(float)*host->nb,             hipMemcpyHostToDevice );

    CUDAMEMCPY( host->node_dev,          &cudaNode_host,     sizeof(CudaNode),                   hipMemcpyHostToDevice );
}

// copy the node statistics from the device to the host.
void CopyNodeFromDevice(Node *host)
{
    CudaNode cudaNode_host;

    // copy struct from device
    CUDAMEMCPY( &cudaNode_host, host->node_dev, sizeof(CudaNode), hipMemcpyDeviceToHost );

    host->winner = cudaNode_host.winner;
    host->clustErr = cudaNode_host.clustErr;

    if( cudaNode_host.inputOffsets == NULL )
    {
        CUDAMEMCPY( host->input, cudaNode_host.input, sizeof(float)*host->ni, hipMemcpyDeviceToHost );
    }

    // copy from pointers given from the struct
    CUDAMEMCPY( host->mu,        cudaNode_host.mu,           sizeof(float)*host->nb*host->ns,    hipMemcpyDeviceToHost );
    CUDAMEMCPY( host->sigma,     cudaNode_host.sigma,        sizeof(float)*host->nb*host->ns,    hipMemcpyDeviceToHost );
    CUDAMEMCPY( host->starv,     cudaNode_host.starv,        sizeof(float)*host->nb,             hipMemcpyDeviceToHost );
    CUDAMEMCPY( host->pBelief,   cudaNode_host.pBelief,      sizeof(float)*host->nb,             hipMemcpyDeviceToHost );
    CUDAMEMCPY( host->beliefEuc, cudaNode_host.beliefEuc,    sizeof(float)*host->nb,             hipMemcpyDeviceToHost );
    CUDAMEMCPY( host->beliefMal, cudaNode_host.beliefMal,    sizeof(float)*host->nb,             hipMemcpyDeviceToHost );
    CUDAMEMCPY( host->pBelief,   cudaNode_host.pBelief,      sizeof(float)*host->nb,             hipMemcpyDeviceToHost );
}

//  CalculateDistances:
//      CUDA kernel that takes in a node and an observation
//        and writes the unnormalized belief to the particular
//        node.
//
//
//              block = (number of nodes) x (max belief dimensionality)
//             thread = (max state dimensionality) x 1
//         shared mem = euclidean sum  (length: ns)
//                      malhanobis sum (length: ns)
__global__ void CalculateDistances( CudaNode *n, float *framePtr )
{
    // shared array -- includes the euclidean and malhanobis arrays for reduction
    extern __shared__ float shared[];
    
    // value for how far the observation (or collection of beliefs from a set of previous
    // nodes) deviates from each centroid
    float delta;

    // pointers in shared memory to the euclidean and malhanobis sums
    float *sumEuc, *sumMal;

    // grab pointer to the node we want to get distances for
    n = &n[blockIdx.x];

    uint i;          // iterator for the reduction
    uint mIdx;       // entry in the mu/sigma matrix to calculate delta
    uint ns;         // node state size
    uint ni;         // node input size
    uint np;         // node parent belief size
    uint nb;         // node belief size

    ns = n->ns;
    ni = n->ni;
    nb = n->nb;
    np = n->np;

    // maxNS is likely greater than n->ns.  don't execute if we're not in range.
    // we do this because the state dimensionalities differ between nodes, but
    // we can't modify the size of the kernel that is called.  if the state
    // dimensionality is out of range, effectively execute a no-op.
    if( threadIdx.x < ns )
    {
        // point euclidean and malhanobis arrays to shared mem
        sumEuc = (float *) &shared[ns*0];
        sumMal = (float *) &shared[ns*1];

        // get entry in the mu/sigma matrices to calculate for this thread
        mIdx = blockIdx.y * ns + threadIdx.x;

        // get difference for each dimension between the input state (input + prev belief) and every
        // centroid location
        if( threadIdx.x < ni )
        {
            if( n->inputOffsets != NULL )
            {
                delta = n->mu[mIdx] - framePtr[n->inputOffsets[threadIdx.x]];
            }
            else
            {
                delta = n->mu[mIdx] - n->input[threadIdx.x];
            }
        }
        else if( threadIdx.x < ni + nb )
        {
            delta = (n->mu[mIdx] - n->pBelief[threadIdx.x-ni]) * LAMBDA;
        }
        else
        {
            if( np > 0 )
            {
                delta = (n->mu[mIdx] - n->parent_pBelief[threadIdx.x-ni-nb]) * GAMMA;
            }
        }

        delta *= n->starv[blockIdx.y];
        delta *= delta;

        syncthreads();

        sumEuc[threadIdx.x] = delta;

        if( n->sigma[mIdx] == 0 )
        {
            sumMal[threadIdx.x] = 0;
        }
        else
        {
            sumMal[threadIdx.x] = delta / n->sigma[mIdx];
        }

        // sync threads before summing up the columns
        syncthreads();

        // reduce euc and mal partial sums.  total sum will be in
        // sumEuc[0] and sumMal[0].

        for( i=1; i < ns; i <<= 1 )
        {
            if( threadIdx.x % (i*2) == 0 && threadIdx.x + i < ns )
            {
                
                sumEuc[threadIdx.x] += sumEuc[threadIdx.x+i];
                sumMal[threadIdx.x] += sumMal[threadIdx.x+i];
            }
            syncthreads();
        }
        

        // get inverse of distance (provides "confidence" or a value of
        // closeness from the centroid to the observation)
        if( threadIdx.x == 0 )
        {
            sumEuc[0] = sqrt(sumEuc[0]);
            sumMal[0] = sqrt(sumMal[0]);

            // save the euclidean distance for cluster err calculation
            n->dist[blockIdx.y] = sumEuc[0];

            if( sumEuc[0] == 0 )
            {
                n->beliefEuc[blockIdx.y] = 1;
            }
            else
            {
                n->beliefEuc[blockIdx.y] = 1 / sumEuc[0];
            }

            if( sumMal[0] == 0 )
            {
                n->beliefMal[blockIdx.y] = 1;
            }
            else
            {
                n->beliefMal[blockIdx.y] = 1 / sumMal[0];
            }
        }
    }
}

// NormalizeBelief:
//      CUDA kernel that normalizes the belief of a node such that all
//      the individual components sum to 1.  This gives each belief a
//      probability that the observation is "close" to a centroid.
//
//                block = individual node
//               thread = belief dimension for a node
//           shared mem = euclidean sum
//                        malhanobis sum

__global__ void NormalizeBelief(CudaNode *n)
{
    extern __shared__ float shared[];
    uint i;

    // grab the node we want
    n = &n[blockIdx.x];

    uint nb;         // number of centroids

    nb = n->nb;

    if( threadIdx.x < nb )
    {
        float *normEuc = (float *) &shared[nb*0];
        float *normMal = (float *) &shared[nb*1];

        // copy normEuc and normMal from global memory
        normEuc[threadIdx.x] = n->beliefEuc[threadIdx.x];
        normMal[threadIdx.x] = n->beliefMal[threadIdx.x];

        // make sure normEuc and normMal are completely populated
        syncthreads();

        // calculate the normalization constant for the belief
        for( i=1; i < nb; i <<= 1 )
        {
            if( threadIdx.x % (i*2) == 0 && threadIdx.x + i < nb )
            {
                normEuc[threadIdx.x] += normEuc[threadIdx.x + i];
                normMal[threadIdx.x] += normMal[threadIdx.x + i];
            }

            syncthreads();
        }

        // normalize the output
        if( normMal[0] == 0 )
        {
            // if the normalization const is 0, the node should
            // have no particular belief.
            n->beliefMal[threadIdx.x] = 1 / nb;
        }
        else
        {
            // otherwise, normalize the sum to 1
            n->beliefMal[threadIdx.x] /= normMal[0];
        }

        // same behavior as malhanobis normalization, see above
        if( normEuc[0] == 0 )
        {
            n->beliefEuc[threadIdx.x] = 1 / nb;
        }
        else
        {
            n->beliefEuc[threadIdx.x] /= normEuc[0];
        }

        // update belief
        n->pBelief[threadIdx.x] = n->beliefMal[threadIdx.x];
    }
}


// NormalizeBeliefGetWinner:
//      CUDA kernel that normalizes the belief (as above) and determines the winning
//        centroid
//
//                block = individual node
//               thread = belief dimension for a node
//           shared mem = euclidean sum
//                        max euclidean belief (to determine the winning centroid)
//                        malhanobis sum
//                        max euclidean belief index (to modify the winning centroid)

__global__ void NormalizeBeliefGetWinner( CudaNode *n )
{
    extern __shared__ float shared[];
    
    uint     i;

    float  *normEuc, *maxEuc, *normMal;
    uint    *maxIdx;

    // pick our particular node
    n = &n[blockIdx.x];

    uint nb;         // number of centroids

    nb = n->nb;

    if( threadIdx.x < nb )
    {
        // set up shared pointers
        normEuc = (float *) &shared[nb*0];
        normMal = (float *) &shared[nb*1];
        maxEuc = (float *) &shared[nb*2];
        maxIdx = (uint *) &shared[nb*3];

        // populate shared memory for reductions
        normEuc[threadIdx.x] = n->beliefEuc[threadIdx.x];
        normMal[threadIdx.x] = n->beliefMal[threadIdx.x];
        maxEuc[threadIdx.x] = n->beliefEuc[threadIdx.x];
        maxIdx[threadIdx.x] = threadIdx.x;

        syncthreads();

        for( i=1; i < nb; i <<= 1 )
        {
            if( threadIdx.x % (i*2) == 0 && threadIdx.x + i < nb )
            {
                // reduce euclidean and malhanobis sums
                normEuc[threadIdx.x] += normEuc[threadIdx.x + i];
                normMal[threadIdx.x] += normMal[threadIdx.x + i];

                // reduce winning centroid
                if( maxEuc[threadIdx.x] < maxEuc[threadIdx.x + i] )
                {
                    maxEuc[threadIdx.x] = maxEuc[threadIdx.x + i];
                    maxIdx[threadIdx.x] = maxIdx[threadIdx.x + i];
                } 
            }

            syncthreads();
        }
        
        // normalize the output
        if( normMal[0] == 0 )
        {
            // if the normalization const is 0, the node should
            // have no particular belief.
            n->beliefMal[threadIdx.x] = 1 / nb;
        }
        else
        {
            // otherwise, normalize the sum to 1
            n->beliefMal[threadIdx.x] /= normMal[0];
        }

        // same behavior as malhanobis normalization, see above
        if( normEuc[0] == 0 )
        {
            n->beliefEuc[threadIdx.x] = 1 / nb;
        }
        else
        {
            n->beliefEuc[threadIdx.x] /= normEuc[0];
        }
    }

    syncthreads();

    if( threadIdx.x == 0 )
    {
        n->clustErr = n->dist[maxIdx[0]];
        n->winner = maxIdx[0];
    }
}

//  UpdateWinner:
//    CUDA Kernel that updates the winning centroid
//
//              block = (number of nodes) x 1
//             thread = (max state dimensionality) x 1
__global__ void UpdateWinner( CudaNode *n, float *framePtr )
{
    // value for how far the observation (or collection of beliefs from a set of previous
    // nodes) deviates from each centroid
    float delta;

    // grab pointer to the node we want to get distances for
    n = &n[blockIdx.x];

    uint mIdx;       // entry in the mu/sigma matrix to calculate delta

    uint nb;         // number of centroids
    uint ns;         // state dimensionality
    uint ni;         // input dimensionality
    uint np;         // parent belief dimensionality
    uint winner;     // winner idx

    nb = n->nb;
    ns = n->ns;
    np = n->np;
    ni = n->ni;
    winner = n->winner;

    // maxNS is likely greater than ns.  don't execute if we're not in range.
    // we do this because the state dimensionalities differ between nodes, but
    // we can't modify the size of the kernel that is called.  if the state
    // dimensionality is out of range, effectively execute a no-op.
    if( threadIdx.x < ns )
    {
        // get entry in the mu/sigma matrices to calculate for this thread
        mIdx = winner * ns + threadIdx.x;

        // get difference for each dimension between the input state (input + prev belief) and every
        // centroid location
        if( threadIdx.x < ni )
        {
            if( n->inputOffsets != NULL )
            {
                delta = n->mu[mIdx] - framePtr[n->inputOffsets[threadIdx.x]];
            }
            else
            {
                delta = n->mu[mIdx] - n->input[threadIdx.x];
            }
        }
        else if( threadIdx.x < ni + nb )
        {
            delta = (n->mu[mIdx] - n->pBelief[threadIdx.x-ni])*LAMBDA;
        }
        else
        {
            if( np > 0 )
            {
                delta = (n->mu[mIdx] - n->parent_pBelief[threadIdx.x-ni-nb]) * GAMMA;
            }
        }
        
        // update mu and sigma
        n->mu[mIdx] -= ALPHA*delta;
        n->sigma[mIdx] -= BETA*(n->sigma[mIdx] - delta*delta);

        // update starv
        if( threadIdx.x < nb )
        {
            n->starv[threadIdx.x] = (1 - STARVCOEFF) * n->starv[threadIdx.x];

            if( threadIdx.x == n->winner )
            {
                n->starv[threadIdx.x] += STARVCOEFF;
            }
        }
    }

    // update belief
    if( threadIdx.x < nb )
    {
        n->pBelief[threadIdx.x] = n->beliefMal[threadIdx.x];
    }
}

// a quick function to print remaining memory on the card (helps to debug any
// memory leak issues)
void cudaPrintMemory()
{
    size_t mfree, mtotal;

    hipMemGetInfo(&mfree, &mtotal);
    printf("Usage: %012zu/%012zu\n", mfree, mtotal);
}
